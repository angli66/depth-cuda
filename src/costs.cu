#include "hip/hip_runtime.h"
/**
    This file is part of sgm. (https://github.com/dhernandez0/sgm).

    Copyright (c) 2016 Daniel Hernandez Juarez.

    sgm is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    sgm is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with sgm.  If not, see <http://www.gnu.org/licenses/>.

	Modified by Ang Li, Jul 2022

**/

#include "costs.h"
#include <stdio.h>

__global__ void 
__launch_bounds__(1024, 2)
CenterSymmetricCensusKernelSM2(const uint8_t *im, const uint8_t *im2, cost_t *transform, cost_t *transform2, const uint32_t rows, const uint32_t cols,
								const uint8_t censusWidth, const uint8_t censusHeight) {
	const int idx = blockIdx.x*blockDim.x+threadIdx.x;
	const int idy = blockIdx.y*blockDim.y+threadIdx.y;

	const int top = (censusHeight-1)/2;
	const int left = (censusWidth-1)/2;
	const int win_cols = (32+left*2);
	const int win_rows = (32+top*2);

	extern __shared__ uint8_t mem[];
	uint8_t *window = mem;
	uint8_t *window2 = mem + win_cols*win_rows;

	const int id = threadIdx.y*blockDim.x+threadIdx.x;
	const int sm_row = id / win_cols;
	const int sm_col = id % win_cols;

	const int im_row = blockIdx.y*blockDim.y+sm_row-top;
	const int im_col = blockIdx.x*blockDim.x+sm_col-left;
	const bool boundaries = (im_row >= 0 && im_col >= 0 && im_row < rows && im_col < cols);
	window[sm_row*win_cols+sm_col] = boundaries ? im[im_row*cols+im_col] : 0;
	window2[sm_row*win_cols+sm_col] = boundaries ? im2[im_row*cols+im_col] : 0;

	// Not enough threads to fill window and window2
	const int block_size = blockDim.x*blockDim.y;
	if(id < (win_cols*win_rows-block_size)) {
		const int id = threadIdx.y*blockDim.x+threadIdx.x+block_size;
		const int sm_row = id / win_cols;
		const int sm_col = id % win_cols;

		const int im_row = blockIdx.y*blockDim.y+sm_row-top;
		const int im_col = blockIdx.x*blockDim.x+sm_col-left;
		const bool boundaries = (im_row >= 0 && im_col >= 0 && im_row < rows && im_col < cols);
		window[sm_row*win_cols+sm_col] = boundaries ? im[im_row*cols+im_col] : 0;
		window2[sm_row*win_cols+sm_col] = boundaries ? im2[im_row*cols+im_col] : 0;
	}

	__syncthreads();
	cost_t census = 0;
	cost_t census2 = 0;
	if(idy < rows && idx < cols) {
			for(int k = 0; k < censusHeight/2; k++) {
				for(int m = 0; m < censusWidth; m++) {
					const uint8_t e1 = window[(threadIdx.y+k)*win_cols+threadIdx.x+m];
					const uint8_t e2 = window[(threadIdx.y+2*top-k)*win_cols+threadIdx.x+2*left-m];
					const uint8_t i1 = window2[(threadIdx.y+k)*win_cols+threadIdx.x+m];
					const uint8_t i2 = window2[(threadIdx.y+2*top-k)*win_cols+threadIdx.x+2*left-m];

					const int shft = k*censusWidth+m;
					// Compare to the center
					cost_t tmp = (e1 >= e2);
					// Shift to the desired position
					tmp <<= shft;
					// Add it to its place
					census |= tmp;
					// Compare to the center
					cost_t tmp2 = (i1 >= i2);
					// Shift to the desired position
					tmp2 <<= shft;
					// Add it to its place
					census2 |= tmp2;
				}
			}
			if(censusHeight % 2 != 0) {
				const int k = censusHeight/2;
				for(int m = 0; m < censusWidth/2; m++) {
					const uint8_t e1 = window[(threadIdx.y+k)*win_cols+threadIdx.x+m];
					const uint8_t e2 = window[(threadIdx.y+2*top-k)*win_cols+threadIdx.x+2*left-m];
					const uint8_t i1 = window2[(threadIdx.y+k)*win_cols+threadIdx.x+m];
					const uint8_t i2 = window2[(threadIdx.y+2*top-k)*win_cols+threadIdx.x+2*left-m];

					const int shft = k*censusWidth+m;
					// Compare to the center
					cost_t tmp = (e1 >= e2);
					// Shift to the desired position
					tmp <<= shft;
					// Add it to its place
					census |= tmp;
					// Compare to the center
					cost_t tmp2 = (i1 >= i2);
					// Shift to the desired position
					tmp2 <<= shft;
					// Add it to its place
					census2 |= tmp2;
				}
			}

		transform[idy*cols+idx] = census;
		transform2[idy*cols+idx] = census2;
	}
}
